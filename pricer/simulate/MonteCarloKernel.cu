#include "hip/hip_runtime.h"
#include <simulate/MonteCarlo.h>
#include <simulate/SumReduction.cuh>
#include <stdio.h>

#define BLOCK_N 256
#define THREAD_N 256

__device__ void randNormal(
    MonteCarlo *plan,
    hiprandState *state,
    double *choMatrix,
    double *depend,
    double *independ)
{
    int size = plan->basketSize;

    for (int i = 0; i < size; i++)
    {
        independ[i] = hiprand_normal(state);
    }

    for (int i = 0; i < size; i++)
    {
        double corNormal = 0;
        for (int j = 0; j < size; j++)
        {
            corNormal += independ[j] * choMatrix[i * size + j];
        }
        depend[i] = corNormal;
    }
}

__device__ void sumRdx(double *s, double *d, double value)
{
    s[threadIdx.x] = value;
    sumReduce<double, THREAD_N, THREAD_N>(s);
    if (threadIdx.x == 0)
    {
        *d = s[0];
    }
}

__device__ double optionValue(MonteCarlo *plan, double value)
{
    return exp(-plan->interest * plan->maturity) * (value > 0 ? value : 0);
}

__global__ void sumReduceKernel(MonteCarlo *plan, double *value, double *sum, double *sum2)
{
    __shared__ double sumThread[THREAD_N];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double sumPerThread = 0, sum2PerThread = 0;
    for (int i = idx; i < plan->pathNum; i += blockDim.x * gridDim.x)
    {
        sumPerThread += value[i];
        sum2PerThread += value[i] * value[i];
    }
    sumRdx(sumThread, &sum[blockIdx.x], sumPerThread);
    sumRdx(sumThread, &sum2[blockIdx.x], sum2PerThread);
}

void MonteCarlo::statisticGPU(MonteCarlo *plan, double *value, double &mean, double &std)
{
    double *sum;
    double *sumHost;
    double *sum2;
    double *sum2Host;

    hipMalloc(&sum, sizeof(double) * BLOCK_N);
    hipMalloc(&sum2, sizeof(double) * BLOCK_N);
    hipHostMalloc(&sumHost, sizeof(double) * BLOCK_N);
    hipHostMalloc(&sum2Host, sizeof(double) * BLOCK_N);

    double sumRes = 0;
    double sum2Res = 0;

    sumReduceKernel<<<BLOCK_N, THREAD_N>>>(plan, value, sum, sum2);

    hipMemcpy(sumHost, sum, BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sum2Host, sum2, BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < BLOCK_N; i++)
    {
        sumRes += sumHost[i];
        sum2Res += sum2Host[i];
    }

    mean = sumRes / pathNum;
    std = std::sqrt(sum2Res / pathNum - mean * mean);

    hipHostFree(sumHost);
    hipHostFree(sum2Host);
    hipFree(sum);
    hipFree(sum2);
}

__global__ void covSumReduceKernel(MonteCarlo *plan, double *a, double *b, double *sum)
{
    __shared__ double sumThread[THREAD_N];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double sumPerThread = 0;
    for (int i = idx; i < plan->pathNum; i += blockDim.x * gridDim.x)
    {
        sumPerThread += a[i] * b[i];
    }
    sumRdx(sumThread, &sum[blockIdx.x], sumPerThread);
}

double MonteCarlo::covarianceGPU(MonteCarlo *plan, double *arith, double *geo, double arithMean, double geoMean)
{
    double *sum;
    double *sumHost;
    hipMalloc(&sum, sizeof(double) * BLOCK_N);
    hipHostMalloc(&sumHost, sizeof(double) * BLOCK_N);

    double sumRes = 0;
    covSumReduceKernel<<<BLOCK_N, THREAD_N>>>(plan, arith, geo, sum);

    hipMemcpy(sumHost, sum, BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < BLOCK_N; i++)
    {
        sumRes += sumHost[i];
    }
    hipHostFree(sumHost);
    hipFree(sum);
    return sumRes / pathNum - arithMean * geoMean;
}

__global__ void variationReduceKernel(MonteCarlo *plan, double *dst, double *arithPayoff, double *geoPayoff, double theta)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < plan->pathNum; i += blockDim.x * gridDim.x)
    {
        dst[i] = arithPayoff[i] + theta * (plan->geoExp - geoPayoff[i]);
    }
}

__global__ void monteCarloOptionKernel(
    MonteCarlo *plan,
    double *choMatrix,
    double *price,
    double *volatility,
    double *drift,
    double *currents,
    double *depend,
    double *independ,
    double *sum,
    double *sumOutput,
    double *sum2,
    double *sum2Output,
    double *sumX,
    double *sumXOutput,
    double *arithPayoff,
    double *geoPayoff)
{
    __shared__ double sumThread[THREAD_N];
    hiprandState state;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = plan->basketSize;
    int offset = idx * size;
    double dt = plan->maturity / plan->observation;

    hiprand_init(1230, idx, 0, &state);
    for (int i = 0; i < size; i++)
    {
        sum[offset + i] = 0;
        sum2[offset + i] = 0;
        sumX[offset + i] = 0;
    }

    for (int i = idx; i < plan->pathNum; i += blockDim.x * gridDim.x)
    {
        double arithMean = 0;
        double geoMean = 1;

        for (int j = 0; j < size; j++)
        {
            currents[offset + j] = price[j];
        }
        for (int j = 0; j < plan->observation; j++)
        {
            randNormal(plan, &state, choMatrix, depend + offset, independ + offset);
            for (int j = 0; j < size; j++)
            {
                double var = depend[offset + j];
                sum[offset + j] += var;
                sum2[offset + j] += var * var;
                for (int k = 0; k < size; k++)
                {
                    double val = depend[offset + k];
                    sumX[offset * size + j * size + k] += var * val;
                }
            }
            for (int k = 0; k < size; k++)
            {
                double growthFactor = drift[k] * exp(volatility[k] * sqrt(dt) * depend[offset + k]);
                currents[offset + k] *= growthFactor;
                arithMean += currents[offset + k];
                geoMean *= currents[offset + k];
            }
        }

        arithMean /= plan->observation * size;
        geoMean = pow(geoMean, 1 / (double)(plan->observation * size));
        if (plan->type == CALL)
        {
            arithPayoff[i] = optionValue(plan, arithMean - plan->strike);
            geoPayoff[i] = optionValue(plan, geoMean - plan->strike);
        }
        else if (plan->type == PUT)
        {
            arithPayoff[i] = optionValue(plan, plan->strike - arithMean);
            geoPayoff[i] = optionValue(plan, plan->strike - geoMean);
        }
    }

    for (int i = 0; i < size; i++)
    {
        sumRdx(sumThread, &sumOutput[blockIdx.x * size + i], sum[offset + i]);
        sumRdx(sumThread, &sum2Output[blockIdx.x * size + i], sum2[offset + i]);

        for (int j = 0; j < size; j++)
        {
            sumRdx(sumThread, &sumXOutput[blockIdx.x * size * size + i * size + j], sumX[offset * size + i * size + j]);
        }
    }
}

Result MonteCarlo::simulateGPU(double *expectation, double *covMatrix)
{
    MonteCarlo *plan;
    double *pChoMatrix;
    double *pPrice;
    double *pVolatility;
    double *pDrift;

    double *currents;
    double *depend;
    double *independ;
    double *sum;
    double *sumOutput;
    double *sumHost;
    double *sum2;
    double *sum2Output;
    double *sum2Host;
    double *sumX;
    double *sumXOutput;
    double *sumXHost;

    double *arithPayoff;
    double *geoPayoff;

    int size = this->basketSize;

    hipMalloc(&plan, sizeof(MonteCarlo));
    hipMalloc(&pChoMatrix, size * size * sizeof(double));
    hipMalloc(&pPrice, size * sizeof(double));
    hipMalloc(&pVolatility, size * sizeof(double));
    hipMalloc(&pDrift, size * sizeof(double));

    int totalThread = BLOCK_N * THREAD_N;

    hipMalloc(&currents, sizeof(double) * size * totalThread);
    hipMalloc(&depend, sizeof(double) * size * totalThread);
    hipMalloc(&independ, sizeof(double) * size * totalThread);
    hipMalloc(&sum, sizeof(double) * size * totalThread);
    hipMalloc(&sum2, sizeof(double) * size * totalThread);
    hipMalloc(&sumX, sizeof(double) * size * size * totalThread);

    hipMalloc(&sumOutput, sizeof(double) * size * BLOCK_N);
    hipMalloc(&sum2Output, sizeof(double) * size * BLOCK_N);
    hipMalloc(&sumXOutput, sizeof(double) * size * size * BLOCK_N);
    hipMalloc(&arithPayoff, this->pathNum * sizeof(double));
    hipMalloc(&geoPayoff, this->pathNum * sizeof(double));

    hipMemcpy(plan, this, sizeof(MonteCarlo), hipMemcpyHostToDevice);
    hipMemcpy(pChoMatrix, this->choMatrix, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(pPrice, this->price, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(pVolatility, this->volatility, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(pDrift, this->drift, size * sizeof(double), hipMemcpyHostToDevice);

    monteCarloOptionKernel<<<BLOCK_N, THREAD_N>>>(
        plan, pChoMatrix,
        pPrice, pVolatility,
        pDrift, currents,
        depend, independ,
        sum, sumOutput,
        sum2, sum2Output,
        sumX, sumXOutput,
        arithPayoff, geoPayoff);

    double aMean, gMean, aStd, gStd;

    statisticGPU(plan, arithPayoff, aMean, aStd);
    statisticGPU(plan, geoPayoff, gMean, gStd);

    hipHostMalloc(&sumHost, sizeof(double) * size * BLOCK_N);
    hipHostMalloc(&sum2Host, sizeof(double) * size * BLOCK_N);
    hipHostMalloc(&sumXHost, sizeof(double) * size * size * BLOCK_N);
    hipMemcpy(sumHost, sumOutput, size * BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sum2Host, sum2Output, size * BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sumXHost, sumXOutput, size * size * BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++)
    {
        expectation[i] = 0;
        for (int j = 0; j < size; j++)
        {
            covMatrix[i * size + j] = 0;
        }
    }
    for (int i = 0; i < BLOCK_N; i++)
    {
        for (int j = 0; j < size; j++)
        {
            expectation[j] += sumHost[i * size + j];
            for (int k = 0; k < size; k++)
            {
                covMatrix[j * size + k] += sumXHost[i * size * size + j * size + k];
            }
        }
    }

    int pathNum = this->pathNum;

    for (int i = 0; i < size; i++)
    {
        expectation[i] /= pathNum;
    }

    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            covMatrix[i * size + j] = covMatrix[i * size + j] / pathNum - expectation[i] * expectation[j];
        }
    }

    Result ret;

    // double stdDev = sqrt(((double)pathNum * pay2Ret - payArithRet * payArithRet) / ((double)pathNum * (double)(pathNum - 1)));
    // ret.confidence = (float)(1.96 * stdDev / sqrt((double)pathNum));

    if (isGeo)
    {
        ret.mean = gMean;
    }
    else
    {
        if (controlVariate)
        {
            double cov = covarianceGPU(plan, arithPayoff, geoPayoff, aMean, gMean);
            double theta = cov / (gStd * gStd);
            double *newArith;
            hipMalloc(&newArith, this->pathNum * sizeof(double));
            variationReduceKernel<<<BLOCK_N, THREAD_N>>>(plan, newArith, arithPayoff, geoPayoff, theta);
            statisticGPU(plan, newArith, aMean, aStd);

            hipFree(newArith);
            ret.mean = aMean;
        }
        else
            ret.mean = aMean;
    }

    ret.arithPayoff = aMean;
    ret.arith2 = aStd;
    ret.geoPayoff = gMean;
    ret.geo2 = gStd;

    hipHostFree(sumHost);
    hipHostFree(sum2Host);

    hipFree(plan);
    hipFree(pChoMatrix);
    hipFree(pPrice);
    hipFree(pVolatility);
    hipFree(pDrift);

    hipFree(currents);
    hipFree(depend);
    hipFree(independ);
    hipFree(sum);
    hipFree(sum2);
    hipFree(sumOutput);
    hipFree(sum2Output);
    hipFree(sumXOutput);
    hipFree(arithPayoff);
    hipFree(geoPayoff);
    return ret;
}
