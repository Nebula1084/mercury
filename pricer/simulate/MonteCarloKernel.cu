#include "hip/hip_runtime.h"
#include <simulate/MonteCarlo.h>
#include <simulate/SumReduction.cuh>
#include <stdio.h>

#define BLOCK_N 256
#define THREAD_N 256

__device__ void randNormal(
    MonteCarlo *plan,
    hiprandState *state,
    double *choMatrix,
    double *depend,
    double *independ)
{
    int size = plan->basketSize;

    for (int i = 0; i < size; i++)
    {
        independ[i] = hiprand_normal(state);
    }

    for (int i = 0; i < size; i++)
    {
        double corNormal = 0;
        for (int j = 0; j < size; j++)
        {
            corNormal += independ[j] * choMatrix[i * size + j];
        }
        depend[i] = corNormal;
    }
}

__device__ void sumRdx(double *s, double *d, double value)
{
    s[threadIdx.x] = value;
    sumReduce<double, THREAD_N, THREAD_N>(s);
    if (threadIdx.x == 0)
    {
        *d = s[0];
    }
}

__device__ double optionValue(MonteCarlo *plan, double value)
{
    return exp(-plan->interest * plan->maturity) * (value > 0 ? value : 0);
}

__global__ void monteCarloOptionKernel(
    MonteCarlo *plan,
    double *choMatrix,
    double *price,
    double *volatility,
    double *drift,
    double *currents,
    double *depend,
    double *independ,
    double *sum,
    double *sumOutput,
    double *sum2,
    double *sum2Output,
    double *sumX,
    double *sumXOutput,
    double *payArith,
    double *payGeo,
    double *pay2)
{
    __shared__ double sumThread[THREAD_N];
    hiprandState state;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = plan->basketSize;
    int offset = idx * size;
    double dt = plan->maturity / plan->observation;
    double arithPayoff = 0;
    double geoPayoff = 0;
    double payArithPerThread = 0;
    double payGeoPerThread = 0;
    double pay2PerThread = 0;

    hiprand_init(1230, idx, 0, &state);
    for (int i = 0; i < size; i++)
    {
        sum[offset + i] = 0;
        sum2[offset + i] = 0;
        sumX[offset + i] = 0;
    }

    for (int i = idx; i < plan->pathNum; i += blockDim.x * gridDim.x)
    {
        double arithMean = 0;
        double geoMean = 1;

        for (int j = 0; j < size; j++)
        {
            currents[offset + j] = price[j];
        }
        for (int j = 0; j < plan->observation; j++)
        {
            randNormal(plan, &state, choMatrix, depend + offset, independ + offset);
            for (int j = 0; j < size; j++)
            {
                double var = depend[offset + j];
                sum[offset + j] += var;
                sum2[offset + j] += var * var;
                for (int k = 0; k < size; k++)
                {
                    double val = depend[offset + k];
                    sumX[offset * size + j * size + k] += var * val;
                }
            }
            for (int k = 0; k < size; k++)
            {
                double growthFactor = drift[k] * exp(volatility[k] * sqrt(dt) * depend[offset + k]);
                currents[offset + k] *= growthFactor;
                arithMean += currents[offset + k];
                geoMean *= currents[offset + k];
            }
        }

        arithMean /= plan->observation * size;
        geoMean = pow(geoMean, 1 / (double)(plan->observation * size));
        if (plan->type == CALL)
        {
            arithPayoff = optionValue(plan, arithMean - plan->strike);
            geoPayoff = optionValue(plan, geoMean - plan->strike);
        }
        else if (plan->type == PUT)
        {
            arithPayoff = optionValue(plan, plan->strike - arithMean);
            geoPayoff = optionValue(plan, plan->strike - geoMean);
        }

        payArithPerThread += arithPayoff;
        payGeoPerThread += geoPayoff;
        pay2PerThread += arithPayoff * arithPayoff;
    }

    sumRdx(sumThread, &payArith[blockIdx.x], payArithPerThread);
    sumRdx(sumThread, &payGeo[blockIdx.x], payGeoPerThread);
    sumRdx(sumThread, &pay2[blockIdx.x], pay2PerThread);

    for (int i = 0; i < size; i++)
    {
        sumRdx(sumThread, &sumOutput[blockIdx.x * size + i], sum[offset + i]);
        sumRdx(sumThread, &sum2Output[blockIdx.x * size + i], sum2[offset + i]);

        for (int j = 0; j < size; j++)
        {
            sumRdx(sumThread, &sumXOutput[blockIdx.x * size * size + i * size + j], sumX[offset * size + i * size + j]);
        }
    }
}

Result MonteCarlo::simulateGPU(double *expectation, double *covMatrix)
{
    MonteCarlo *plan;
    double *pChoMatrix;
    double *pPrice;
    double *pVolatility;
    double *pDrift;

    double *currents;
    double *depend;
    double *independ;
    double *sum;
    double *sumOutput;
    double *sumHost;
    double *sum2;
    double *sum2Output;
    double *sum2Host;
    double *sumX;
    double *sumXOutput;
    double *sumXHost;

    double *payArith;
    double *payArithHost;
    double *payGeo;
    double *payGeoHost;
    double *pay2;
    double *pay2Host;

    int size = this->basketSize;

    hipMalloc(&plan, sizeof(MonteCarlo));
    hipMalloc(&pChoMatrix, size * size * sizeof(double));
    hipMalloc(&pPrice, size * sizeof(double));
    hipMalloc(&pVolatility, size * sizeof(double));
    hipMalloc(&pDrift, size * sizeof(double));

    int totalThread = BLOCK_N * THREAD_N;

    hipMalloc(&currents, sizeof(double) * size * totalThread);
    hipMalloc(&depend, sizeof(double) * size * totalThread);
    hipMalloc(&independ, sizeof(double) * size * totalThread);
    hipMalloc(&sum, sizeof(double) * size * totalThread);
    hipMalloc(&sum2, sizeof(double) * size * totalThread);
    hipMalloc(&sumX, sizeof(double) * size * size * totalThread);

    hipMalloc(&payArith, sizeof(double) * BLOCK_N);
    hipMalloc(&payGeo, sizeof(double) * BLOCK_N);
    hipMalloc(&pay2, sizeof(double) * BLOCK_N);
    hipMalloc(&sumOutput, sizeof(double) * size * BLOCK_N);
    hipMalloc(&sum2Output, sizeof(double) * size * BLOCK_N);
    hipMalloc(&sumXOutput, sizeof(double) * size * size * BLOCK_N);

    hipMemcpy(plan, this, sizeof(MonteCarlo), hipMemcpyHostToDevice);
    hipMemcpy(pChoMatrix, this->choMatrix, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(pPrice, this->price, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(pVolatility, this->volatility, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(pDrift, this->drift, size * sizeof(double), hipMemcpyHostToDevice);

    monteCarloOptionKernel<<<BLOCK_N, THREAD_N>>>(
        plan, pChoMatrix,
        pPrice, pVolatility,
        pDrift, currents,
        depend, independ,
        sum, sumOutput,
        sum2, sum2Output,
        sumX, sumXOutput,
        payArith, payGeo, pay2);

    hipHostMalloc(&payArithHost, sizeof(double) * BLOCK_N);
    hipHostMalloc(&payGeoHost, sizeof(double) * BLOCK_N);
    hipHostMalloc(&pay2Host, sizeof(double) * BLOCK_N);
    hipHostMalloc(&sumHost, sizeof(double) * size * BLOCK_N);
    hipHostMalloc(&sum2Host, sizeof(double) * size * BLOCK_N);
    hipHostMalloc(&sumXHost, sizeof(double) * size * size * BLOCK_N);
    hipMemcpy(payArithHost, payArith, BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(payGeoHost, payGeo, BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(pay2Host, pay2, BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sumHost, sumOutput, size * BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sum2Host, sum2Output, size * BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sumXHost, sumXOutput, size * size * BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);

    double payArithRet = 0;
    double payGeoRet = 0;
    double pay2Ret = 0;

    for (int i = 0; i < size; i++)
    {
        expectation[i] = 0;
        for (int j = 0; j < size; j++)
        {
            covMatrix[i * size + j] = 0;
        }
    }
    for (int i = 0; i < BLOCK_N; i++)
    {
        payArithRet += payArithHost[i];
        payGeoRet += payGeoHost[i];
        pay2Ret += pay2Host[i];
        for (int j = 0; j < size; j++)
        {
            expectation[j] += sumHost[i * size + j];
            for (int k = 0; k < size; k++)
            {
                covMatrix[j * size + k] += sumXHost[i * size * size + j * size + k];
            }
        }
    }

    int pathNum = this->pathNum;

    for (int i = 0; i < size; i++)
    {
        expectation[i] /= pathNum;
    }

    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            covMatrix[i * size + j] = covMatrix[i * size + j] / pathNum - expectation[i] * expectation[j];
        }
    }

    Result ret;

    ret.expected = payArithRet / (double)pathNum;
    ret.arithPayoff = payArithRet / (double)pathNum;
    ret.geoPayoff = payGeoRet / (double)pathNum;
    double stdDev = sqrt(((double)pathNum * pay2Ret - payArithRet * payArithRet) / ((double)pathNum * (double)(pathNum - 1)));
    ret.confidence = (float)(1.96 * stdDev / sqrt((double)pathNum));

    hipHostFree(sumHost);
    hipHostFree(sum2Host);
    hipHostFree(payArithHost);
    hipHostFree(payGeoHost);
    hipHostFree(pay2Host);

    hipFree(plan);
    hipFree(pChoMatrix);
    hipFree(pPrice);
    hipFree(pVolatility);
    hipFree(pDrift);

    hipFree(currents);
    hipFree(depend);
    hipFree(independ);
    hipFree(sum);
    hipFree(sum2);
    hipFree(sumOutput);
    hipFree(sum2Output);
    hipFree(sumXOutput);
    hipFree(payArith);
    hipFree(payGeo);
    hipFree(pay2);
    return ret;
}
