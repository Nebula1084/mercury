#include "hip/hip_runtime.h"
#include <american/American.h>

#include <stdio.h>

#define BLOCK_N 256
#define THREAD_N 256

__global__ void expiryValueKernel(American *plan, double *value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int step = plan->step;
    int strike = plan->instrument.strike;

    for (int i = idx; i <= step; i += blockDim.x * gridDim.x)
    {
        double d = plan->asset.price * exp(plan->vDt * (2.0f * i - step));
        if (plan->instrument.type == CALL)
            d = d - strike;
        else if (plan->instrument.type == PUT)
            d = strike - d;
        value[i] = (d > 0) ? d : 0;
    }
}

__global__ void binomialKernel(American *plan, int iter, double *value, double *next)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < iter; i += blockDim.x * gridDim.x)
    {
        next[i] = plan->puByDf * value[i + 1] + plan->pdByDf * value[i];
    }
}

double American::binomialGPU()
{
    American *plan;
    double *value, *next;
    double result;

    hipMalloc(&plan, sizeof(American));
    hipMalloc(&value, sizeof(double) * (step + 1));
    hipMalloc(&next, sizeof(double) * (step + 1));

    hipMemcpy(plan, this, sizeof(American), hipMemcpyHostToDevice);

    expiryValueKernel<<<BLOCK_N, THREAD_N>>>(plan, value);
    for (int i = step; i > 0; i--)
    {
        double *tmp;
        binomialKernel<<<BLOCK_N, THREAD_N>>>(plan, i, value, next);
        tmp = value;
        value = next;
        next = tmp;
    }

    hipMemcpy(&result, value, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(plan);
    hipFree(value);
    hipFree(next);

    return result;
}
