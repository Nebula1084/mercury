#include "hip/hip_runtime.h"
#include <asian/MonteCarlo.h>
#include <asian/SumReduction.cuh>

#define BLOCK_N 256
#define THREAD_N 256

__device__ void randNormal(
    Asian *asian,
    hiprandState *state,
    double *choMatrix,
    double *depend,
    double *independ)
{
    int size = asian->basketSize;

    for (int i = 0; i < size; i++)
    {
        independ[i] = hiprand_normal(state);
    }

    for (int i = 0; i < size; i++)
    {
        double corNormal = 0;
        for (int j = 0; j < size; j++)
        {
            corNormal += independ[j] * choMatrix[i * size + j];
        }
        depend[i] = corNormal;
    }
}

__global__ void monteCarloOptionKernel(
    Asian *asian,
    double *choMatrix,
    double *depend,
    double *independ,
    double *sum,
    double *sumOutput,
    double *sum2,
    double *sum2Output)
{
    __shared__ double sumThread[THREAD_N];
    double sumPerThread = 0;
    hiprandState state;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = asian->basketSize;
    int offset = idx * size;

    hiprand_init(1234, idx, 0, &state);
    for (int i = 0; i < size; i++)
    {
        sum[offset + i] = 0;
        sum2[offset + i] = 0;
    }

    for (int i = idx; i < asian->pathNum; i += blockDim.x * gridDim.x)
    {
        // randNormal(asian, &state, choMatrix, depend + offset, independ + offset);

        for (int j = 0; j < size; j++)
        {
            // double var = depend[offset] + j;
            double var = 1;
            sum[offset + j] += var;
            sum2[offset + j] += var * var;
        }
        sumPerThread += 1;
    }    

    for (int i = 0; i < size; i++)
    {
        sumThread[threadIdx.x] = sum[offset + i];
        sumReduce<double, THREAD_N, THREAD_N>(sumThread);
        if (threadIdx.x == 0)
        {
            sumOutput[blockIdx.x * size + i] = sumThread[0];
        }
        sumThread[threadIdx.x] = sum2[offset + i];
        sumReduce<double, THREAD_N, THREAD_N>(sumThread);
        if (threadIdx.x == 0)
        {
            sum2Output[blockIdx.x * size + i] = sumThread[0];
        }
    }
}

double monteCarloGPU(Asian *asian)
{
    Asian *option;
    double *choMatrix;

    double *depend;
    double *independ;
    double *sum;
    double *sumOutput;
    double *sumHost;
    double *sum2;
    double *sum2Output;
    double *sum2Host;
    double ret = 0;
    double ret2 = 0;

    int size = asian->basketSize;

    hipMalloc(&option, sizeof(Asian));
    hipMalloc(&choMatrix, size * size * sizeof(double));

    int totalThread = BLOCK_N * THREAD_N;
    hipMalloc(&depend, sizeof(double) * size * totalThread);
    hipMalloc(&independ, sizeof(double) * size * totalThread);
    hipMalloc(&sum, sizeof(double) * size * totalThread);
    hipMalloc(&sum2, sizeof(double) * size * totalThread);

    hipMalloc(&sumOutput, sizeof(double) * size * BLOCK_N);
    hipMalloc(&sum2Output, sizeof(double) * size * BLOCK_N);
    hipMemcpy(option, asian, sizeof(Asian), hipMemcpyHostToDevice);
    hipMemcpy(choMatrix, asian->choMatrix, size * size * sizeof(double), hipMemcpyHostToDevice);

    monteCarloOptionKernel<<<BLOCK_N, THREAD_N>>>(option, choMatrix, depend, independ, sum, sumOutput, sum2, sum2Output);
    
    hipHostMalloc(&sumHost, sizeof(double) * size * BLOCK_N);
    hipHostMalloc(&sum2Host, sizeof(double) * size * BLOCK_N);
    hipMemcpy(sumHost, sumOutput, size *BLOCK_N* sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sum2Host, sum2Output, size * BLOCK_N* sizeof(double), hipMemcpyDeviceToHost);
    for (int i=0; i<BLOCK_N; i++){
        ret += sumHost[i*size];
        ret2 += sum2Host[i*size];
    }

    printf("total:%f\n", ret);
    double mean = ret / asian->pathNum;
    
    printf("%f %f\n", mean, ret2 / asian->pathNum - mean * mean);
    hipHostFree(sumHost);
    hipHostFree(sum2Host);

    hipFree(choMatrix);
    hipFree(asian);
    hipFree(depend);
    hipFree(independ);
    hipFree(sum);
    hipFree(sum2);
    hipFree(sumOutput);
    hipFree(sum2Output);
    return 0;
}
