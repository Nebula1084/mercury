#include "hip/hip_runtime.h"
#include <asian/MonteCarlo.h>
#include <asian/SumReduction.cuh>

#define BLOCK_N 256
#define THREAD_N 256

__device__ void randNormal(
    Asian *asian,
    hiprandState *state,
    double *choMatrix,
    double *depend,
    double *independ)
{
    int size = asian->basketSize;

    for (int i = 0; i < size; i++)
    {
        independ[i] = hiprand_normal(state);
    }

    for (int i = 0; i < size; i++)
    {
        double corNormal = 0;
        for (int j = 0; j < size; j++)
        {
            corNormal += independ[j] * choMatrix[i * size + j];
        }
        depend[i] = corNormal;
    }
}

__global__ void monteCarloOptionKernel(
    Asian *asian,
    double *choMatrix,
    double *depend,
    double *independ,
    double *sum,
    double *sumOutput,
    double *sum2,
    double *sum2Output,
    double *sumX,
    double *sumXOutput)
{
    __shared__ double sumThread[THREAD_N];
    hiprandState state;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = asian->basketSize;
    int offset = idx * size;

    hiprand_init(1230, idx, 0, &state);
    for (int i = 0; i < size; i++)
    {
        sum[offset + i] = 0;
        sum2[offset + i] = 0;
        sumX[offset + i] = 0;
    }

    for (int i = idx; i < asian->pathNum; i += blockDim.x * gridDim.x)
    {
        randNormal(asian, &state, choMatrix, depend + offset, independ + offset);

        for (int j = 0; j < size; j++)
        {
            double var = depend[offset + j];
            sum[offset + j] += var;
            sum2[offset + j] += var * var;
            for (int k = 0; k < size; k++)
            {
                double val = depend[offset + k];
                sumX[offset * size + j * size + k] += var * val;
            }
        }
    }

    for (int i = 0; i < size; i++)
    {
        sumThread[threadIdx.x] = sum[offset + i];
        sumReduce<double, THREAD_N, THREAD_N>(sumThread);
        if (threadIdx.x == 0)
        {
            sumOutput[blockIdx.x * size + i] = sumThread[0];
        }
        sumThread[threadIdx.x] = sum2[offset + i];
        sumReduce<double, THREAD_N, THREAD_N>(sumThread);
        if (threadIdx.x == 0)
        {
            sum2Output[blockIdx.x * size + i] = sumThread[0];
        }
        for (int j = 0; j < size; j++)
        {
            sumThread[threadIdx.x] = sumX[offset * size + i * size + j];
            sumReduce<double, THREAD_N, THREAD_N>(sumThread);
            if (threadIdx.x == 0)
            {
                sumXOutput[blockIdx.x * size * size + i * size + j] = sumThread[0];
            }
        }
    }
}

double monteCarloGPU(Asian *asian, double *expectation, double *covMatrix)
{
    Asian *option;
    double *choMatrix;

    double *depend;
    double *independ;
    double *sum;
    double *sumOutput;
    double *sumHost;
    double *sum2;
    double *sum2Output;
    double *sum2Host;
    double *sumX;
    double *sumXOutput;
    double *sumXHost;

    int size = asian->basketSize;

    hipMalloc(&option, sizeof(Asian));
    hipMalloc(&choMatrix, size * size * sizeof(double));

    int totalThread = BLOCK_N * THREAD_N;
    hipMalloc(&depend, sizeof(double) * size * totalThread);
    hipMalloc(&independ, sizeof(double) * size * totalThread);
    hipMalloc(&sum, sizeof(double) * size * totalThread);
    hipMalloc(&sum2, sizeof(double) * size * totalThread);
    hipMalloc(&sumX, sizeof(double) * size * size * totalThread);

    hipMalloc(&sumOutput, sizeof(double) * size * BLOCK_N);
    hipMalloc(&sum2Output, sizeof(double) * size * BLOCK_N);
    hipMalloc(&sumXOutput, sizeof(double) * size * size * BLOCK_N);

    hipMemcpy(option, asian, sizeof(Asian), hipMemcpyHostToDevice);
    hipMemcpy(choMatrix, asian->choMatrix, size * size * sizeof(double), hipMemcpyHostToDevice);

    monteCarloOptionKernel<<<BLOCK_N, THREAD_N>>>(
        option, choMatrix,
        depend, independ,
        sum, sumOutput,
        sum2, sum2Output,
        sumX, sumXOutput);

    hipHostMalloc(&sumHost, sizeof(double) * size * BLOCK_N);
    hipHostMalloc(&sum2Host, sizeof(double) * size * BLOCK_N);
    hipHostMalloc(&sumXHost, sizeof(double) * size * size * BLOCK_N);
    hipMemcpy(sumHost, sumOutput, size * BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sum2Host, sum2Output, size * BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sumXHost, sumXOutput, size * size * BLOCK_N * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++)
    {
        expectation[i] = 0;
        for (int j = 0; j < size; j++)
        {
            covMatrix[i * size + j] = 0;
        }
    }
    for (int i = 0; i < BLOCK_N; i++)
    {
        for (int j = 0; j < size; j++)
        {
            expectation[j] += sumHost[i * size + j];
            for (int k = 0; k < size; k++)
            {
                covMatrix[j * size + k] += sumXHost[i * size * size + j * size + k];
            }
        }
    }

    for (int i = 0; i < size; i++)
    {
        expectation[i] /= asian->pathNum;
    }

    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            covMatrix[i * size + j] = covMatrix[i * size + j] / asian->pathNum - expectation[i] * expectation[j];
        }
    }

    hipHostFree(sumHost);
    hipHostFree(sum2Host);

    hipFree(choMatrix);
    hipFree(asian);
    hipFree(depend);
    hipFree(independ);
    hipFree(sum);
    hipFree(sum2);
    hipFree(sumOutput);
    hipFree(sum2Output);
    hipFree(sumXOutput);
    return 0;
}
